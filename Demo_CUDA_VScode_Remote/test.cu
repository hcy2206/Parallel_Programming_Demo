#include <iostream>
#include <hip/hip_runtime.h>

__global__ void kernel(int *a) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    a[i] = i;
}

int main() {
    int n = 1024;
    int *a;
    hipMallocManaged(&a, n * sizeof(int));
    kernel<<<n/256, 256>>>(a);
    hipDeviceSynchronize();
    for (int i = 0; i < n; ++i) {
        std::cout << a[i] << " ";
    }
    std::cout << std::endl;
    hipFree(a);
    return 0;
}